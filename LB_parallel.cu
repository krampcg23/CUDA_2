#include "hip/hip_runtime.h"
/* CSCI 563 Programming Assignment 3
   Clayton Kramp
*/

#include <stdio.h>
#include <math.h>
#include <assert.h>

#define THREADS_PER_LINE 16
using namespace std;


__global__ void loadBalancedSpMV(float* t, float* b, int* ptr, float* data, int* ind, int n) {


    int myi = blockIdx.x * blockDim.x + threadIdx.x;

    int lb = ptr[myi / THREADS_PER_LINE];
    int ub = ptr[(myi / THREADS_PER_LINE) + 1];
    extern __shared__ float partialSum[];

    partialSum[threadIdx.x] = 0;
    partialSum[threadIdx.x + THREADS_PER_LINE] = 0;
    for (int j = lb + threadIdx.x; j < ub; j += THREADS_PER_LINE) {
        int index = ind[j];
        partialSum[threadIdx.x] += data[j] * b[index];
    }

    for (unsigned int stride = THREADS_PER_LINE; stride > 0; stride /= 2) {
         __syncthreads();
          if (threadIdx.x < stride)
               partialSum[threadIdx.x] += partialSum[threadIdx.x+stride];
    }
    if (threadIdx.x == 0)
        t[myi / THREADS_PER_LINE] = partialSum[threadIdx.x];
}

main (int argc, char **argv) {
  FILE *fp;
  char line[1024]; 
  int *ptr, *indices;
  float *data, *b, *t;
  int i,j;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix

  // Open input file and read to end of comments
  if (argc !=2) abort(); 

  if ((fp = fopen(argv[1], "r")) == NULL) {
    abort();
  }

  fgets(line, 128, fp);
  while (line[0] == '%') {
    fgets(line, 128, fp); 
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;
  for (i=0; i<n; i++) {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));  
    indices[i]--;  // start numbering at 0
    if (r!=lastr) { 
      ptr[r-1] = i; 
      lastr = r; 
    }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data  
  for (i=0; i<nr; i++) {
    t[i] = 0.0;
  }

  for (i=0; i<nc; i++) {
    b[i] = (float) rand()/1111111111;
  }

    
  // TODO: Compute result on GPU and compare output
  float* deviceT;
  hipMalloc(&deviceT, nr * sizeof(float));
  hipMemcpy(deviceT, t, nr * sizeof(float), hipMemcpyHostToDevice);

  float* deviceB;
  hipMalloc(&deviceB, nc * sizeof(float));
  hipMemcpy(deviceB, b, nc * sizeof(float), hipMemcpyHostToDevice);

  int* devicePtr;
  hipMalloc(&devicePtr, (nr+1) * sizeof(int));
  hipMemcpy(devicePtr, ptr, (nr+1) * sizeof(int), hipMemcpyHostToDevice);

  float* deviceData;
  hipMalloc(&deviceData, n * sizeof(float));
  hipMemcpy(deviceData, data, n * sizeof(float), hipMemcpyHostToDevice);

  int* deviceIndices;
  hipMalloc(&deviceIndices, n * sizeof(int));
  hipMemcpy(deviceIndices, indices, n * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16,1,1);
  dim3 numBlocks(nr, 1, 1);
  loadBalancedSpMV<<<numBlocks, threadsPerBlock, nc+THREADS_PER_LINE>>>(deviceT, deviceB, devicePtr, deviceData, deviceIndices, n);

  float* newT = (float *) malloc(nr*sizeof(float));
  hipMemcpy(newT, deviceT, nr*sizeof(float), hipMemcpyDeviceToHost);

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) {                                                      
    for (j = ptr[i]; j<ptr[i+1]; j++) {
      t[i] = t[i] + data[j] * b[indices[j]];
    }
  }

  for (int i = 0; i < nr; i++) {
      assert(abs(newT[i] - t[i] < 0.0001));
  }
  printf("%d\n", nr);

  hipFree(deviceT);
  hipFree(deviceIndices);
  hipFree(devicePtr);
  hipFree(deviceData);
  hipFree(deviceB);


}
